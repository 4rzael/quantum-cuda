#include "hip/hip_runtime.h"
/**
 * @Author: Nicolas Jankovic <nj203>
 * @Date:   2018-06-16T10:08:10+01:00
 * @Email:  nicolas.jankovic@epitech.eu
 * @Project: CUDA-Based Simulator of Quantum Systems
 * @Filename: QGPU.cuh
 * @Last modified by:   l3ninj
 * @Last modified time: 2018-07-05T14:32:05+01:00
 * @License: MIT License
 */

#include "GPUExecutor.cuh"

GPUExecutor::GPUExecutor()
  : cgpu_() {}


GPUExecutor::~GPUExecutor() = default;


Tvcplxd* GPUExecutor::add(Tvcplxd* a, Tvcplxd* b) {
  Tvcplxd* ptr;

  this->cgpu_.initThrustHostVec((*a), (*b), QCUDA::Vectors::ALL_VECTORS);
  // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_B);
  this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_B);
  ptr = this->cgpu_.performAddOnGPU();
  return (ptr);
}

// Naive CPU implementation, whereas GPU is still in development
Tvcplxd* GPUExecutor::dot(Tvcplxd* a, Tvcplxd* b, int ma, int mb, int na, int nb) {
  Tvcplxd* result = new Tvcplxd(na * mb);

  for (int i = 0; i < na; i++) {
    for (int j = 0; j < mb; j++) {
      (*result)[i * mb + j] = 0;
      for (int k = 0; k < nb; k++) {
        (*result)[i * mb + j] += (*a)[i * ma + k] * (*b)[k * mb + j];
      }
    }
  }
  return result;
  // Tvcplxd* ptr;

  // this->cgpu_.initThrustHostVec((*a), (*b), QCUDA::Vectors::ALL_VECTORS);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_B);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_B);
  // ptr = this->cgpu_.performDotOnGPU(ma, mb, na, nb);
  // return (ptr);
}


// Naive CPU implementation, whereas GPU is still in development
Tvcplxd* GPUExecutor::kron(Tvcplxd* a, Tvcplxd* b, int ma, int mb) {
  int na = a->size() / ma;
  int nb = b->size() / mb;

  Tvcplxd* result = new Tvcplxd(ma * mb * na * nb);

  for (int j = 0; j < na * nb; j++) {
    for (int i = 0; i < ma * mb; i++) {
      (*result)[i + j * ma * mb] = (*b)[i % mb + (j % nb) * mb] *
      (*a)[i / mb + (j / nb) * ma];
    }
  }
  return result;
  // Tvcplxd* ptr;

  // this->cgpu_.initThrustHostVec((*a), (*b), QCUDA::Vectors::ALL_VECTORS);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_B);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_B);
  // ptr = this->cgpu_.performKronOnGPU(a->size() / ma, b->size() / mb, ma, mb);
  // return (ptr);
}


std::complex<double> GPUExecutor::trace(Tvcplxd* a, int m) {
  this->cgpu_.initThrustHostVec((*a), (*a), QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  return(this->cgpu_.performTraceOnGPU(m));
}


// Naive CPU implementation, whereas GPU is still in development
Tvcplxd* GPUExecutor::transpose(Tvcplxd* a, int m, int n) {
  Tvcplxd* result = new Tvcplxd(m * n);

  for(int j = 0; j < n; j++) {
    for (int i = 0; i < m; i++) {
      (*result)[i * n + j] = (*a)[j * m + i];
    }
  }
  return result;
  // Tvcplxd* ptr;

  // this->cgpu_.initThrustHostVec((*a), (*a), QCUDA::Vectors::VECTOR_A);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  // ptr = this->cgpu_.performTransposeOnGPU(m, n);
  // return (ptr);
}


// Naive CPU implementation, whereas GPU is still in development
Tvcplxd* GPUExecutor::normalize(Tvcplxd* a) {
  Tvcplxd* result = new Tvcplxd(a->size());
  std::complex<double> sum = 0;

  for (uint i = 0; i < a->size(); i++) {
    sum += (*a)[i] * (*a)[i];
  }
  if (sum == std::complex<double>(0)) {
    sum = 1;
  }
  sum = sqrt(sum);
  for (uint j = 0; j < a->size(); j++) {
    (*result)[j] = (*a)[j] / sum;
  }
  return result;
  // Tvcplxd* ptr;

  // this->cgpu_.initThrustHostVec((*a), (*a), QCUDA::Vectors::VECTOR_A);
  // // this->cgpu_.assignHostToDevice(QCUDA::Vectors::VECTOR_A);
  // this->cgpu_.convertDeviceToCUDAType(QCUDA::Vectors::VECTOR_A);
  // ptr = this->cgpu_.performNormalizeOnGPU();
  // return (ptr);
}
