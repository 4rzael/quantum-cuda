/**
 * @Author: Nicolas Jankovic <nj203>
 * @Date:   2018-06-16T10:08:10+01:00
 * @Email:  nicolas.jankovic@epitech.eu
 * @Project: CUDA-Based Simulator of Quantum Systems
 * @Filename: QGPU.cuh
 * @Last modified by:   nj203
 * @Last modified time: 2018-06-27T14:51:52+01:00
 * @License: MIT License
 */

#include "QGPU.cuh"
