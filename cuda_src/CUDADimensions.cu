#include "hip/hip_runtime.h"
/**
 * @Author: Nicolas Jankovic <nj203>
 * @Date:   2018-06-16T10:08:10+01:00
 * @Email:  nicolas.jankovic@epitech.eu
 * @Project: CUDA-Based Simulator of Quantum Systems
 * @Filename: CUDADimensions.cu
 * @Last modified by:   nj203
 * @Last modified time: 2018-06-27T14:51:52+01:00
 * @License: MIT License
 */

#include <iostream>
#include <climits>
#include <cmath>

#include "CUDADimensions.cuh"

__host__
QCUDA::CUDADim::CUDADim()
  : gridDim_(dim3(1, 1, 1)),
    blockDim_(dim3(1, 1, 1))
{}


__host__
QCUDA::CUDADim::~CUDADim() = default;


// __host__
// bool	QCUDA::CUDADim::checkDim(QCUDA::) const noexcept {
//   switch
// }


__host__
constexpr void	QCUDA::CUDADim::resetDimensions() noexcept {
  this->gridDim_.x = 1;
  this->gridDim_.y = 1;
  this->gridDim_.z = 1;

  this->blockDim_.x = 1;
  this->blockDim_.y = 1;
  this->blockDim_.z = 1;
}


//init as linear
__host__
void	QCUDA::CUDADim::naiveInit(const hipDeviceProp_t& prop, // CHANGE NAME
				  int nSteps) {
  this->resetDimensions();
  if ((this->gridDim_.x = ((nSteps + (prop.maxThreadsDim[0] - 1)) / prop.maxThreadsDim[0])) == 0) {
    this->gridDim_.x = 1;
  }
  if (this->gridDim_.x > INT_MAX) {
    throw std::runtime_error("The allocation of threads for the run has "
			     "outclassed the maximum numbers of threads in X "
			     "dimension !");
  }
  this->blockDim_.x = prop.maxThreadsDim[0];
}


//init as plan
__host__
void	QCUDA::CUDADim::initForDotProduct(const hipDeviceProp_t& prop,
					  int m,
					  int n) {
  this->resetDimensions();
  if (m != 1) { // matrix
    const int threadPerDim = min((int)sqrt(prop.maxThreadsPerBlock), min(prop.maxThreadsDim[0], prop.maxThreadsDim[1]));
    this->blockDim_.x = threadPerDim;
    this->blockDim_.y = threadPerDim;
  } else { // vector
    this->blockDim_.x = 1;
    this->blockDim_.y = min(prop.maxThreadsPerBlock, prop.maxThreadsDim[1]);
  }
  if ((this->gridDim_.x = m / blockDim_.x) == 0) {
    this->gridDim_.x = 1;
  }
  if ((this->gridDim_.y = n / blockDim_.y) == 0) {
    this->gridDim_.y = 1;
  }
}


__host__
void	QCUDA::CUDADim::initGridAndBlock(const hipDeviceProp_t& prop,
					 QCUDA::QOperation&& op,
					 int m,
					 int n) {
  switch (op) {
  case QCUDA::QOperation::DOT:
    this->initForDotProduct(prop, m, n);
    break;
  case QCUDA::QOperation::KRONECKER:
    this->initForDotProduct(prop, m, n);
    break;
  case QCUDA::QOperation::TRANSPOSE:
    this->initForDotProduct(prop, m, n);
    break;
  case QCUDA::QOperation::NORMALIZE:
    this->initForDotProduct(prop, m, n);
    break;
  default:
    this->naiveInit(prop, m);
    break;
  }
  std::cout << "this->gridDim_.x: " << this->gridDim_.x << std::endl;
  std::cout << "this->gridDim_.y: " << this->gridDim_.y << std::endl;
  std::cout << "this->blockDim_.x: " << this->blockDim_.x << std::endl;
  std::cout << "this->blockDim_.y: " << this->blockDim_.y << std::endl;
}

__host__
const dim3&	QCUDA::CUDADim::getGridDim() const {
  return (this->gridDim_);
}

__host__
const dim3&	QCUDA::CUDADim::getBlockDim() const {
  return (this->blockDim_);
}
